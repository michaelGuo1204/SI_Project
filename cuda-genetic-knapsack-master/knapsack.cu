#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand/hiprand_kernel.h"
#include "utils.h"
#include "knapsack.h"

extern "C"
{
    __global__
    void initChromosomes(char* chromosomes, int length, hiprandState* states)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        hiprandState localState;
        char localPart;
        hiprand_init(SEED, id, 0, &localState);
        for(int i = 0; i < length; ++i)
        {
            if(hiprand_uniform(&localState) < INIT_RATE)
                localPart ^= (1<<(i % BYTE_SIZE));
            if(i % BYTE_SIZE == BYTE_SIZE-1)
                chromosomes[(id * length / BYTE_SIZE) + (i / BYTE_SIZE)] = localPart;
        }
        states[id] = localState;
    }

    __global__
    void computeFitness(char* chromosomes, int length, int* values, int* weights, int knapsackSize, int* fitness)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        char localPart;
        int w = 0, v = 0;
        for(int i = 0; i < length; ++i)
        {
            if(i % BYTE_SIZE == 0)
                localPart = chromosomes[(id * length / BYTE_SIZE) + (i / BYTE_SIZE)];
            if(localPart & (1<< (i%BYTE_SIZE)))
            {
                w += weights[i];
                v += values[i];
            }
        }
        if(w > knapsackSize)
            v = 0;
        fitness[id] = v;
    }

    __global__
    void prefixSumInBlocks(int* array, int* sumsInBlocks)
    {
        __shared__ int part[BLOCK_WIDTH];
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        part[threadIdx.x] = array[id];
        
        __syncthreads();
        for(int len = 1; len < blockDim.x; len *= 2)
        {
            int tmp = 0;
            if (threadIdx.x >= len)
                tmp = part[threadIdx.x - len];
            __syncthreads();
            part[threadIdx.x] += tmp;
            __syncthreads();
        }

        array[id] = part[threadIdx.x];
        if(threadIdx.x == blockDim.x-1)
            sumsInBlocks[blockIdx.x] = part[threadIdx.x];
    }

    __global__
    void propagatePrefixSums(int* array, int* sumsInBlocks)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        if(blockIdx.x > 0)
        {
            int localSum = sumsInBlocks[blockIdx.x-1];
            array[id] += localSum;
        }
    }

    __global__
    void selectChromosomes(char* currentPop, char* nextPop, int length, int* fitness, hiprandState* states)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        int rnd = int(hiprand_uniform(&states[id]) * (fitness[POPULATION_SIZE-1] + 0.999999));
        int beg = 0, end = POPULATION_SIZE-1, mid;
        while(beg < end)
        {
            mid = (beg + end) / 2;
            if(fitness[mid] < rnd)
                beg = mid+1;
            else
                end = mid;
        }
        __syncthreads();

        for(int i = 0; i < length / BYTE_SIZE; ++i)
            nextPop[(id * length / BYTE_SIZE) + i] = currentPop[(beg * length / BYTE_SIZE) + i];
    }

    __global__
    void crossOverChromosomes(char* chromosomes, int length, hiprandState* states)
    {
        int id = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
        int place = int(hiprand_uniform(&states[id]) * (length + 0.999999));
        if(place == length || hiprand_uniform(&states[id]) > CROSSING_OVER_RATE)
            return;

        char tmp, tmp2, mask;
        mask = (1 << (place % BYTE_SIZE)) - 1;
        tmp = chromosomes[(id * length / BYTE_SIZE) + place / BYTE_SIZE];
        tmp2 = chromosomes[((id+1) * length / BYTE_SIZE) + place / BYTE_SIZE];
        chromosomes[(id * length / BYTE_SIZE) + place / BYTE_SIZE] = (tmp & mask) | (tmp2 & ~mask);
        chromosomes[((id+1) * length / BYTE_SIZE) + place / BYTE_SIZE] = (tmp2 & mask) | (tmp & ~mask);;

        for(int i = place / BYTE_SIZE + 1; i < length / BYTE_SIZE; ++i)
        {
            tmp = chromosomes[(id * length / BYTE_SIZE) + i];
            chromosomes[(id * length / BYTE_SIZE) + i] = chromosomes[((id+1) * length / BYTE_SIZE) + i];
            chromosomes[((id+1) * length / BYTE_SIZE) + i] = tmp;
        }
    }

    __global__
    void mutateChromosomes(char* chromosomes, int length, hiprandState* states)
    {
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        hiprandState localState = states[id];
        char localPart;
        for(int i = 0; i < length; ++i)
        {
            if(i % BYTE_SIZE == 0)
                localPart = chromosomes[(id * length / BYTE_SIZE) + (i / BYTE_SIZE)];
            if(hiprand_uniform(&localState) < MUTATION_RATE)
                localPart ^= (1<<(i % BYTE_SIZE));
            if(i % BYTE_SIZE == BYTE_SIZE-1)
                chromosomes[(id * length / BYTE_SIZE) + (i / BYTE_SIZE)] = localPart;
        }
        states[id] = localState;
    }

    __global__
    void findBestChromosome(char* chromosomes, int* fitness, int* results, int* where)
    {
        __shared__ int maxes[BLOCK_WIDTH];
        __shared__ int indices[BLOCK_WIDTH];
        int id = blockIdx.x * blockDim.x + threadIdx.x;
        maxes[threadIdx.x] = fitness[id];
        indices[threadIdx.x] = id;
        
        __syncthreads();
        for(int len = 1; len < blockDim.x; len *= 2)
        {
            int otherValue = 0, otherIndex;
            if (threadIdx.x % (2*len) == 0 && threadIdx.x + len < blockDim.x)
            {
                otherValue = maxes[threadIdx.x + len];
                otherIndex = indices[threadIdx.x + len];
            }
            __syncthreads();
            if(otherValue > maxes[threadIdx.x])
            {
                maxes[threadIdx.x] = otherValue;
                indices[threadIdx.x] = otherIndex;
            }
            __syncthreads();
        }

        if(threadIdx.x == 0)
        {
            results[blockIdx.x] = maxes[0];
            where[blockIdx.x] = indices[0];
        }

    }
}
